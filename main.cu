
#include <hip/hip_runtime.h>
#include "iostream"

#define N 257

__global__ void sum_of_array(float *arr1, float *arr2, float *arr3) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    arr3[i] = arr1[i] + arr2[i];
}

void initialize_array(float *arr, int size) {
    for (int i = 0; i < size; i++) {
        arr[i] = (float) random();
    }
}

int main() {
    float *arr1, *arr2, *arr3, *d_arr1, *d_arr2, *d_arr3;
    size_t n_byte = N * sizeof(float);

    arr1 = (float *) malloc(n_byte);
    arr2 = (float *) malloc(n_byte);
    arr3 = (float *) malloc(n_byte);

    initialize_array(arr1, N);
    initialize_array(arr2, N);
    initialize_array(arr3, N);

    printf("start cudaMalloc\n");
    hipMalloc((void **) &d_arr1, N);
    hipMalloc((void **) &d_arr2, N);
    hipMalloc((void **) &d_arr3, N);
    printf("finish cudaMalloc\n");

    printf("start cudaMemcpy\n");
    hipMemcpy(d_arr1, arr1, n_byte, hipMemcpyHostToDevice);
    hipMemcpy(d_arr2, arr2, n_byte, hipMemcpyHostToDevice);
    hipMemcpy(d_arr3, arr3, n_byte, hipMemcpyHostToDevice);
    printf("finish cudaMemcpy\n");
    printf("start kernel function\n");
    sum_of_array<<<(N + 255) / 256, 256>>>(d_arr1, d_arr2, d_arr3);
    printf("finish kernel function\n");
    hipMemcpy(arr3, d_arr3, n_byte, hipMemcpyDeviceToHost);
    printf("%f", *arr3);

}
